#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "freshman.h"


void sumArrays(float * a,float * b,float * res,const int size)
{
  for(int i=0;i<size;i+=4)
  {
    res[i]=a[i]+b[i];
    res[i+1]=a[i+1]+b[i+1];
    res[i+2]=a[i+2]+b[i+2];
    res[i+3]=a[i+3]+b[i+3];
  }
}
__global__ void sumArraysGPU(float*a,float*b,float*res)
{
  //int i=threadIdx.x;
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  // blockIdx.x: 当前线程所在的block在grid中的索引（从0开始）
  // blockDim.x: 每个block中的线程数量（在你的代码中是1024）。
  // threadIdx.x: 当前线程在block内的局部索引（0到1023）。
  // 得到的i是当前线程的全局唯一索引
  res[i]=a[i]+b[i];
  printf("[%d]\t\tres: %f\n", i, res[i]);
}
// Main function with command line arguments
int main(int argc, char **argv) 
{
    // Set up CUDA device
    int dev = 0;  // Use device 0 (the first GPU)
    hipSetDevice(dev);  // Set the current CUDA device

    // Define vector size and calculate memory requirements
    int nElem = 1 << 14;  // Vector size = 2^14 = 16384 elements
    printf("Vector size: %d\n", nElem);
    int nByte = sizeof(float) * nElem;  // Total bytes needed for each array

    // Allocate host (CPU) memory
    float *a_h = (float*)malloc(nByte);       // Host array a
    float *b_h = (float*)malloc(nByte);       // Host array b
    float *res_h = (float*)malloc(nByte);     // Host result from CPU computation
    float *res_from_gpu_h = (float*)malloc(nByte);  // Host result from GPU computation
    
    // Initialize result arrays to 0
    memset(res_h, 0, nByte);
    memset(res_from_gpu_h, 0, nByte);

    // Allocate device (GPU) memory
    float *a_d, *b_d, *res_d;
    CHECK(hipMalloc((float**)&a_d, nByte));    // Device array a
    CHECK(hipMalloc((float**)&b_d, nByte));    // Device array b
    CHECK(hipMalloc((float**)&res_d, nByte));  // Device result array

    // Initialize input data on host
    initialData(a_h, nElem);  // Fill array a with initial values
    initialData(b_h, nElem);  // Fill array b with initial values

    // Copy data from host -> device
    CHECK(hipMemcpy(a_d, a_h, nByte, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d, b_h, nByte, hipMemcpyHostToDevice));

    // Define CUDA kernel execution configuration
    dim3 block(1024);  // Each block has 1024 threads
    dim3 grid(nElem / block.x);  // Calculate number of blocks needed
    
    // Launch the kernel to perform vector addition on GPU
    sumArraysGPU<<<grid, block>>>(a_d, b_d, res_d);
    printf("Execution configuration<<<%d, %d>>>\n", grid.x, block.x);

    // Copy results back from device to host
    CHECK(hipMemcpy(res_from_gpu_h, res_d, nByte, hipMemcpyDeviceToHost));
    
    // Perform the same computation on CPU for verification
    sumArrays(a_h, b_h, res_h, nElem);

    // Compare GPU and CPU results
    checkResult(res_h, res_from_gpu_h, nElem);

    // Free device memory
    hipFree(a_d);
    hipFree(b_d);
    hipFree(res_d);

    // Free host memory
    free(a_h);
    free(b_h);
    free(res_h);
    free(res_from_gpu_h);

    return 0;
}
